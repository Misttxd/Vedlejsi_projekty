#include "hip/hip_runtime.h"
// ***********************************************************************
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
// Image stacking and part transformations.
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "cuda_img.h"



__global__ void kernel_combine(CudaImg output, CudaImg img1, CudaImg img2) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= output.m_size.x || y >= output.m_size.y) return;
    
    if (x < img1.m_size.x) {
        output.m_p_uchar4[y * output.m_size.x + x] = img1.m_p_uchar4[y * img1.m_size.x + x];
    } else {
        int x2 = x - img1.m_size.x;
        if (x2 < img2.m_size.x) {
            output.m_p_uchar4[y * output.m_size.x + x] = img2.m_p_uchar4[y * img2.m_size.x + x2];
        }
    }
}

__global__ void kernel_overlay(CudaImg base, CudaImg overlay) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= base.m_size.x || y >= base.m_size.y) return;
    
    uchar4 base_pixel = base.m_p_uchar4[y * base.m_size.x + x];
    uchar4 overlay_pixel = overlay.m_p_uchar4[y * overlay.m_size.x + x];
    
    float alpha = 0.5f;
    base_pixel.x = base_pixel.x * (1 - alpha) + overlay_pixel.x * alpha;
    base_pixel.y = base_pixel.y * (1 - alpha) + overlay_pixel.y * alpha;
    base_pixel.z = base_pixel.z * (1 - alpha) + overlay_pixel.z * alpha;
    
    base.m_p_uchar4[y * base.m_size.x + x] = base_pixel;
}




void cu_combine_images(CudaImg output, CudaImg img1, CudaImg img2) {
    dim3 block(16, 16);
    dim3 grid((output.m_size.x + block.x - 1) / block.x, 
              (output.m_size.y + block.y - 1) / block.y);
    kernel_combine<<<grid, block>>>(output, img1, img2);
    hipDeviceSynchronize();
}

void cu_overlay_image(CudaImg base, CudaImg overlay) {
    dim3 block(16, 16);
    dim3 grid((base.m_size.x + block.x - 1) / block.x, 
              (base.m_size.y + block.y - 1) / block.y);
    kernel_overlay<<<grid, block>>>(base, overlay);
    hipDeviceSynchronize();
}
