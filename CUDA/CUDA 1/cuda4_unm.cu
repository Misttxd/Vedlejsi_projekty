#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_redukce(CudaImg t_color_cuda_img, float r_remove, float g_remove, float b_remove)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ];

    l_bgr.x = l_bgr.x * (1.0f - b_remove); // Blue
    l_bgr.y = l_bgr.y * (1.0f - g_remove); // Green
    l_bgr.z = l_bgr.z * (1.0f - r_remove); // Red

    // Store BW point to new image
    t_color_cuda_img.m_p_uchar3[l_y * t_color_cuda_img.m_size.x + l_x] = l_bgr;
}

void redukace(CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img, float r_remove, float g_remove, float b_remove)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_redukce<<< l_blocks, l_threads >>>( t_color_cuda_img, r_remove, g_remove, b_remove);

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}
