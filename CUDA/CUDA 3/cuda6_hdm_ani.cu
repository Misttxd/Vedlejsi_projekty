#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"
#include "animation.h"

__global__ void kernel_creategradient( CudaImg t_color_cuda_img )
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    int l_dy = l_x * t_color_cuda_img.m_size.y / t_color_cuda_img.m_size.x + l_y - t_color_cuda_img.m_size.y;
    unsigned char l_color = 255 * abs( l_dy ) / t_color_cuda_img.m_size.y;

    uchar3 l_bgr = ( l_dy < 0 ) ? ( uchar3 ) { l_color, 255 - l_color, 0 } : ( uchar3 ) { 0, 255 - l_color, l_color };

    t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = l_bgr;
}


__global__ void kernel_insertimage(CudaImg t_big_cuda_img, CudaImg t_small_cuda_img, int2 t_position)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_small_cuda_img.m_size.y) return;
    if (l_x >= t_small_cuda_img.m_size.x) return;

    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    
    if (l_by >= t_big_cuda_img.m_size.y || l_by < 0) return;
    if (l_bx >= t_big_cuda_img.m_size.x || l_bx < 0) return;

    uchar4 l_fg_bgra = t_small_cuda_img.m_p_uchar4[l_y * t_small_cuda_img.m_size.x + l_x];
    uchar3 l_bg_bgr = t_big_cuda_img.m_p_uchar3[l_by * t_big_cuda_img.m_size.x + l_bx];
    uchar3 l_bgr = {0, 0, 0};

    l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * (255 - l_fg_bgra.w) / 255;
    l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * (255 - l_fg_bgra.w) / 255;
    l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * (255 - l_fg_bgra.w) / 255;

    t_big_cuda_img.m_p_uchar3[l_by * t_big_cuda_img.m_size.x + l_bx] = l_bgr;
}

void cu_insertimage(CudaImg t_big_cuda_img, CudaImg t_small_cuda_img, int2 t_position)
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks((t_small_cuda_img.m_size.x + l_block_size - 1) / l_block_size,
                 (t_small_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_insertimage<<<l_blocks, l_threads>>>(t_big_cuda_img, t_small_cuda_img, t_position);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}


__global__ void kernel_bilin_scale(CudaImg orig, CudaImg resize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= resize.m_size.x || y >= resize.m_size.y) return;
    
    float scale_x = (orig.m_size.x - 1) / (float)resize.m_size.x;
    float scale_y = (orig.m_size.y - 1) / (float)resize.m_size.y;
    
    float orig_x = x * scale_x;
    float orig_y = y * scale_y;
    
    float diff_x = orig_x - (int)orig_x;
    float diff_y = orig_y - (int)orig_y;
    
    int x0 = (int)orig_x;
    int y0 = (int)orig_y;
    int x1 = min(x0 + 1, orig.m_size.x - 1);
    int y1 = min(y0 + 1, orig.m_size.y - 1);
    
    uchar4 p00 = orig.m_p_uchar4[y0 * orig.m_size.x + x0];
    uchar4 p01 = orig.m_p_uchar4[y0 * orig.m_size.x + x1];
    uchar4 p10 = orig.m_p_uchar4[y1 * orig.m_size.x + x0];
    uchar4 p11 = orig.m_p_uchar4[y1 * orig.m_size.x + x1];
    
    uchar4 result;
    result.x = p00.x * (1 - diff_y) * (1 - diff_x) +
               p01.x * (1 - diff_y) * diff_x +
               p10.x * diff_y * (1 - diff_x) +
               p11.x * diff_y * diff_x;
    result.y = p00.y * (1 - diff_y) * (1 - diff_x) +
               p01.y * (1 - diff_y) * diff_x +
               p10.y * diff_y * (1 - diff_x) +
               p11.y * diff_y * diff_x;
    result.z = p00.z * (1 - diff_y) * (1 - diff_x) +
               p01.z * (1 - diff_y) * diff_x +
               p10.z * diff_y * (1 - diff_x) +
               p11.z * diff_y * diff_x;
    result.w = p00.w * (1 - diff_y) * (1 - diff_x) +
               p01.w * (1 - diff_y) * diff_x +
               p10.w * diff_y * (1 - diff_x) +
               p11.w * diff_y * diff_x;
    
    resize.m_p_uchar4[y * resize.m_size.x + x] = result;
}

void cu_bilin_scale(CudaImg orig, CudaImg resize) {
    dim3 block(16, 16);
    dim3 grid((resize.m_size.x + block.x - 1) / block.x,
              (resize.m_size.y + block.y - 1) / block.y);
    kernel_bilin_scale<<<grid, block>>>(orig, resize);
    hipDeviceSynchronize();
}

__global__ void kernel_rotate(CudaImg t_orig, CudaImg t_rotate, float t_alpha)
{
    int l_rotate_x = blockIdx.x * blockDim.x + threadIdx.x;
    int l_rotate_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (l_rotate_x >= t_rotate.m_size.x || l_rotate_y >= t_rotate.m_size.y) return;
    
    float t_sin = sinf(t_alpha);
    float t_cos = cosf(t_alpha);

    int l_crotate_x = l_rotate_x - t_rotate.m_size.x / 2;
    int l_crotate_y = l_rotate_y - t_rotate.m_size.y / 2;

    float l_corig_x = t_cos * l_crotate_x - t_sin * l_crotate_y;
    float l_corig_y = t_sin * l_crotate_x + t_cos * l_crotate_y;
    
    int l_orig_x = l_corig_x + t_orig.m_size.x / 2;
    int l_orig_y = l_corig_y + t_orig.m_size.y / 2;
    
    if (l_orig_x < 0 || l_orig_x >= t_orig.m_size.x) return;
    if (l_orig_y < 0 || l_orig_y >= t_orig.m_size.y) return;

    t_rotate.m_p_uchar4[l_rotate_y * t_rotate.m_size.x + l_rotate_x] = 
        t_orig.m_p_uchar4[l_orig_y * t_orig.m_size.x + l_orig_x];
}

void cu_rotate(CudaImg orig, CudaImg rotate, float alpha) {
    dim3 block(16, 16);
    dim3 grid((rotate.m_size.x + block.x - 1) / block.x, (rotate.m_size.y + block.y - 1) / block.y);
    kernel_rotate<<<grid, block>>>(orig, rotate, alpha);
    hipDeviceSynchronize();
}

void Animation::start(CudaImg t_bg_cuda_img, CudaImg t_ins_cuda_img, CudaImg t_helicopter_cuda_img)
{
    if (m_initialized) return;
    hipError_t l_cerr;

    m_bg_cuda_img = t_bg_cuda_img;
    m_res_cuda_img = t_bg_cuda_img;
    m_ins_cuda_img = t_ins_cuda_img;
    m_helicopter_cuda_img = t_helicopter_cuda_img; 

    l_cerr = hipMalloc(&m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMalloc(&m_ins_cuda_img.m_p_void, m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMalloc(&m_helicopter_cuda_img.m_p_void, m_helicopter_cuda_img.m_size.x * m_helicopter_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMalloc(&m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    m_scaled_cuda_img.m_size.x = m_ins_cuda_img.m_size.x * 2;
    m_scaled_cuda_img.m_size.y = m_ins_cuda_img.m_size.y * 2;
    l_cerr = hipMalloc(&m_scaled_cuda_img.m_p_void, m_scaled_cuda_img.m_size.x * m_scaled_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    m_rotated_cuda_img.m_size.x = m_ins_cuda_img.m_size.x * 3;
    m_rotated_cuda_img.m_size.y = m_ins_cuda_img.m_size.y * 3;
    l_cerr = hipMalloc(&m_rotated_cuda_img.m_p_void, m_rotated_cuda_img.m_size.x * m_rotated_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMemcpy(m_bg_cuda_img.m_p_void, t_bg_cuda_img.m_p_void, 
                       m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3), 
                       hipMemcpyHostToDevice);
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMemcpy(m_ins_cuda_img.m_p_void, t_ins_cuda_img.m_p_void, 
                       m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof(uchar4), 
                       hipMemcpyHostToDevice);
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    l_cerr = hipMemcpy(m_helicopter_cuda_img.m_p_void, t_helicopter_cuda_img.m_p_void, 
                       m_helicopter_cuda_img.m_size.x * m_helicopter_cuda_img.m_size.y * sizeof(uchar4), 
                       hipMemcpyHostToDevice);
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    m_initialized = 1;
}

void Animation::next(CudaImg t_res_pic, int2 t_ball_position, float t_ball_scale, float t_ball_rotation, int2 t_helicopter_position)
{
    if (!m_initialized) return;

    hipError_t cerr;

    cerr = hipMemcpy(m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, 
                     m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3), 
                     hipMemcpyDeviceToDevice);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    cu_insertimage(m_res_cuda_img, m_helicopter_cuda_img, t_helicopter_position);

    if (t_ball_position.x >= 0 && t_ball_position.y >= 0) {
        CudaImg scaled_img = m_scaled_cuda_img;

        float scale_with_safety = t_ball_scale * 1.5f;
        scaled_img.m_size.x = (int)(m_ins_cuda_img.m_size.x * scale_with_safety);
        scaled_img.m_size.y = (int)(m_ins_cuda_img.m_size.y * scale_with_safety);

        cu_bilin_scale(m_ins_cuda_img, scaled_img);


        CudaImg rotated_img = m_rotated_cuda_img;
        rotated_img.m_size = scaled_img.m_size;


        cerr = hipMemset(rotated_img.m_p_void, 0, 
                        rotated_img.m_size.x * rotated_img.m_size.y * sizeof(uchar4));
        if (cerr != hipSuccess)
            printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));


        cu_rotate(scaled_img, rotated_img, t_ball_rotation);


        int offset_x = (int)((scale_with_safety - t_ball_scale) * m_ins_cuda_img.m_size.x / 2);
        int offset_y = (int)((scale_with_safety - t_ball_scale) * m_ins_cuda_img.m_size.y / 2);
        int2 adjusted_position = {t_ball_position.x - offset_x, t_ball_position.y - offset_y};
        
        cu_insertimage(m_res_cuda_img, rotated_img, adjusted_position);
    }


    cerr = hipMemcpy(t_res_pic.m_p_void, m_res_cuda_img.m_p_void, 
                     m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3), 
                     hipMemcpyDeviceToHost);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));
}


void Animation::stop()
{
    if (!m_initialized) return;

    hipFree(m_bg_cuda_img.m_p_void);
    hipFree(m_res_cuda_img.m_p_void);
    hipFree(m_ins_cuda_img.m_p_void);
    hipFree(m_helicopter_cuda_img.m_p_void); 
    hipFree(m_scaled_cuda_img.m_p_void);
    hipFree(m_rotated_cuda_img.m_p_void); 

    m_initialized = 0;
}